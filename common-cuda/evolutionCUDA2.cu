#include "hip/hip_runtime.h"

#include "evolutionCUDA.h"
#include "cumath.h"

// #define IDX2F(i,j,ld) ((((j)-1)*(ld))+((i)-1)) 1-based
// #define IDX2C(i,j,ld) (((j)*(ld))+(i)) 0-based

struct RadialCoordinates
{ 
  double dr;
  double mass;
  int n;
};

__constant__ double dt;
__constant__ RadialCoordinates r1_dev;
__constant__ RadialCoordinates r2_dev;

inline int number_of_blocks(const int n_threads, const int n)
{ return n/n_threads*n_threads == n ? n/n_threads : n/n_threads+1; }

inline void copy_radial_coordinates_to_device(const RadialCoordinates &r, const int &n, 
					      const double &dr, const double &mass)	    
{									
  RadialCoordinates r_h;
  r_h.dr = dr;
  r_h.mass = mass;
  r_h.n = n;
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(r), &r_h, sizeof(RadialCoordinates))); 
}

static __global__ void setup_momentum_for_fft(double *p, const int n, const double xl)
{ cumath::setup_momentum_for_fft(p, n, xl); }

static __device__ void setup_kinetic_energy_for_fft(double *kin, const int n, const double xl, const double mass)
{ cumath::setup_kinetic_energy_for_fft(kin, n, xl, mass); }

static __global__ void _evolution_with_potential_dt_(Complex *psi, const double *pot, int n)
{
  const int j = threadIdx.x + blockDim.x*blockIdx.x;
  if(j < n) psi[j] *= exp(Complex(0.0, -dt)*pot[j]);
}

static __global__ void _psi_times_kinitic_energy_(Complex *psiOut, const Complex *psiIn, 
						  const double *kin1, const int n1,
						  const double *kin2, const int n2)
{
  extern __shared__ double kin_share[];

  double *sKin1 = (double *) kin_share;
  double *sKin2 = (double *) &sKin1[n1];
  
  setup_kinetic_energy_for_fft(sKin1, r1_dev.n, (r1_dev.n*r1_dev.dr), r1_dev.mass);
  setup_kinetic_energy_for_fft(sKin2, r2_dev.n, (r2_dev.n*r2_dev.dr), r2_dev.mass);
  
  __syncthreads();
  
  const int index = threadIdx.x + blockDim.x*blockIdx.x;
  if(index < n1*n2) {
    int j = index/n1;
    int i = index - j*n1;
    psiOut[index] = psiIn[index]*(sKin1[i] + sKin2[j]);
  }
}

void gpu_memory_usage()
{
  size_t free_byte ;
  size_t total_byte ;
  checkCudaErrors(hipMemGetInfo(&free_byte, &total_byte));
  
  cout << " GPU memory usage:" 
       << " used = " << (total_byte-free_byte)/1024.0/1024.0 << "MB,"
       << " free = " << free_byte/1024.0/1024.0 << "MB,"
       << " total = " << total_byte/1024.0/1024.0 << "MB" <<endl;
}

void EvolutionCUDA::allocate_device_memories()
{ 
  cout << " Allocate device memory" << endl;
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;

  const int n = n1*n2*n_theta;
  
  cout << n1 << " " << n2 << " " << n_theta << " " << n << endl;
  
  // time step
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dt), &time.time_step, sizeof(double)));
  
  if(!pot_dev) {
    checkCudaErrors(hipMalloc(&pot_dev, n*sizeof(double)));
    insist(pot);
    checkCudaErrors(hipMemcpy(pot_dev, pot, n*sizeof(double), hipMemcpyHostToDevice));
  }

  if(!psi_dev) {
    checkCudaErrors(hipMalloc(&psi_dev, n*sizeof(Complex)));
    insist(psi);
    checkCudaErrors(hipMemcpy(psi_dev, psi, n*sizeof(Complex), hipMemcpyHostToDevice));
  }
  
  if(!work_dev) {
    const int max_dim = n1*n2 + n_theta + 1024;
    checkCudaErrors(hipMalloc(&work_dev, max_dim*sizeof(Complex)));
  }
  
  if(!w_dev) {
    checkCudaErrors(hipMalloc(&w_dev, n_theta*sizeof(double)));
    const double *w = theta.w;
    insist(w);
    checkCudaErrors(hipMemcpy(w_dev, w, n_theta*sizeof(double), hipMemcpyHostToDevice));
  }

  if(!kinetic_1_dev) {
    checkCudaErrors(hipMalloc(&kinetic_1_dev, n1*sizeof(double)));
    checkCudaErrors(hipMemcpy(kinetic_1_dev, r1.psq2m,  n1*sizeof(double), hipMemcpyHostToDevice));
  }
  
  if(!kinetic_2_dev) {
    checkCudaErrors(hipMalloc(&kinetic_2_dev, n2*sizeof(double)));
    checkCudaErrors(hipMemcpy(kinetic_2_dev, r2.psq2m,  n2*sizeof(double), hipMemcpyHostToDevice));
  }
  
  copy_radial_coordinates_to_device(r1_dev, r1.n, r1.dr, r1.mass);
  copy_radial_coordinates_to_device(r2_dev, r2.n, r2.dr, r2.mass);

  setup_cublas_handle();
}

void EvolutionCUDA::deallocate_device_memories()
{
  cout << " Deallocate device memory" << endl;

#define _CUDA_FREE_(x) if(x) { checkCudaErrors(hipFree(x)); x = 0; }

  _CUDA_FREE_(pot_dev);
  _CUDA_FREE_(psi_dev);
  _CUDA_FREE_(work_dev);
  _CUDA_FREE_(w_dev);
  _CUDA_FREE_(exp_ipot_dt_dev);
  _CUDA_FREE_(legendre_dev);
  _CUDA_FREE_(weight_legendre_dev);
  _CUDA_FREE_(legendre_psi_dev);
  _CUDA_FREE_(kinetic_1_dev);
  _CUDA_FREE_(kinetic_2_dev);
  
#undef _CUDA_FREE_

  destroy_cublas_handle();
  destroy_cufft_plan_for_psi();
  destroy_cufft_plan_for_legendre_psi();
}

void EvolutionCUDA::setup_cublas_handle()
{
  if(has_cublas_handle) return;
  insist(hipblasCreate(&cublas_handle) == HIPBLAS_STATUS_SUCCESS);
  has_cublas_handle = 1;
}

void EvolutionCUDA::destroy_cublas_handle()
{
  if(!has_cublas_handle) return;
  insist(hipblasDestroy(cublas_handle) == HIPBLAS_STATUS_SUCCESS);
  has_cublas_handle = 0;
}

void EvolutionCUDA::setup_cufft_plan_for_psi()
{
  if(has_cufft_plan_for_psi) return;

  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;
  
  int dim[] = { n1, n2 };

  insist(hipfftPlanMany(&cufft_plan_for_psi, 2, dim, NULL, 1, n1*n2, NULL, 1, n1*n2,
		       HIPFFT_Z2Z, n_theta) == HIPFFT_SUCCESS);
  
  has_cufft_plan_for_psi = 1;
}

void EvolutionCUDA::destroy_cufft_plan_for_psi()
{
  if(!has_cufft_plan_for_psi) return;
  insist(hipfftDestroy(cufft_plan_for_psi) == HIPFFT_SUCCESS);
  has_cufft_plan_for_psi = 0;
}

void EvolutionCUDA::setup_cufft_plan_for_legendre_psi()
{
  if(has_cufft_plan_for_legendre_psi) return;
  
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int m = theta.m + 1;
  
  int dim[] = { n1, n2 };
  
  insist(hipfftPlanMany(&cufft_plan_for_legendre_psi, 2, dim, NULL, 1, n1*n2, NULL, 1, n1*n2,
		       HIPFFT_Z2Z, m) == HIPFFT_SUCCESS);
  
  has_cufft_plan_for_legendre_psi = 1;
}

void EvolutionCUDA::destroy_cufft_plan_for_legendre_psi()
{
  if(!has_cufft_plan_for_legendre_psi) return;
  insist(hipfftDestroy(cufft_plan_for_legendre_psi) == HIPFFT_SUCCESS);
  has_cufft_plan_for_legendre_psi = 0;
}

void EvolutionCUDA::setup_legendre()
{
  if(legendre_dev) return;

  const int &n_theta = theta.n;
  const int m = theta.m + 1;
  const RMat &P = theta.legendre;

  Mat<Complex> P_complex(m, n_theta);
  for(int k = 0; k < n_theta; k++) {
    for(int l = 0; l < m; l++) {
      P_complex(l,k) = Complex(P(l,k), 0.0);
    }
  }
  
  checkCudaErrors(hipMalloc(&legendre_dev, m*n_theta*sizeof(Complex)));

  checkCudaErrors(hipMemcpy(legendre_dev, (const Complex *) P_complex,
			     m*n_theta*sizeof(Complex), hipMemcpyHostToDevice));
}

void EvolutionCUDA::setup_weight_legendre()
{ 
  if(weight_legendre_dev) return;

  const int &n_theta = theta.n;
  const int m = theta.m + 1;
  
  Mat<Complex> weight_legendre(n_theta, m);
  
  const double *w = theta.w;
  const RMat &P = theta.legendre;

  Mat<Complex> &wp = weight_legendre;
  
  for(int l = 0; l < m; l++) {
    const double f = l+0.5;
    for(int k = 0; k < n_theta; k++) {
      wp(k,l) = Complex(f*w[k]*P(l,k), 0.0);
    }
  }
  
  checkCudaErrors(hipMalloc(&weight_legendre_dev, m*n_theta*sizeof(Complex)));
  
  checkCudaErrors(hipMemcpy(weight_legendre_dev, (const Complex *) weight_legendre,
			     m*n_theta*sizeof(Complex), hipMemcpyHostToDevice));
  
}

void EvolutionCUDA::evolution_with_potential_dt()
{
  insist(pot_dev && psi_dev);
  
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;
  const int n = n1*n2*n_theta;
  
  const int n_threads = 1024;
  const int n_blocks = number_of_blocks(n_threads, n);
  
  _evolution_with_potential_dt_<<<n_blocks, n_threads>>>(psi_dev, pot_dev, n);
}

double EvolutionCUDA::potential_energy()
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;

  const double *w = theta.w;
  
  insist(work_dev);
  hipDoubleComplex *psi_tmp_dev = (hipDoubleComplex *) work_dev;
  
  const int n_threads = 1024;
  const int n_blocks = number_of_blocks(n_threads, n1*n2);
  
  double sum = 0.0;
  for(int k = 0; k < n_theta; k++) {
    const hipDoubleComplex *psi_in_dev = (hipDoubleComplex *) psi_dev + k*n1*n2;
    
    cumath::_vector_multiplication_<Complex, Complex, double><<<n_blocks, n_threads>>>
      ((Complex *) psi_tmp_dev, (const Complex *) psi_in_dev, pot_dev+k*n1*n2, n1*n2);
    
    checkCudaErrors(hipDeviceSynchronize());
    
    Complex dot(0.0, 0.0);
    insist(hipblasZdotc(cublas_handle, n1*n2, psi_in_dev, 1, psi_tmp_dev, 1, (hipDoubleComplex *) &dot)
	   == HIPBLAS_STATUS_SUCCESS);
    
    sum += w[k]*dot.real();
  }

  sum *= r1.dr*r2.dr;
  return sum;
}

double EvolutionCUDA::module_for_psi() const
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;
  
  const double *w = theta.w;
  
  double sum= 0.0;
  for(int k = 0; k < n_theta; k++) {
    Complex dot(0.0, 0.0);
    const hipDoubleComplex *psi_ = (hipDoubleComplex *) psi_dev + k*n1*n2;
    insist(hipblasZdotc(cublas_handle, n1*n2, psi_, 1, psi_, 1, (hipDoubleComplex *) &dot)
	   == HIPBLAS_STATUS_SUCCESS);
    sum += w[k]*dot.real();
  }

  sum *= r1.dr*r2.dr;
  return sum;
}

void EvolutionCUDA::setup_legendre_psi()
{
  if(legendre_psi_dev) return;

  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int m = theta.m + 1;
  
  checkCudaErrors(hipMalloc(&legendre_psi_dev, n1*n2*m*sizeof(Complex)));
  checkCudaErrors(hipMemset(legendre_psi_dev, 0, n1*n2*m*sizeof(Complex)));
}

void EvolutionCUDA::cuda_fft_test()
{ 
  cout << " === EvolutionCUDA test ===" << endl;

  insist(psi_dev);
  
  StopWatchInterface *timer = 0;
  sdkCreateTimer(&timer);
  
  const int &total_steps = time.total_steps;
  
  for(int k = 0; k < total_steps; k++) {
    
    cout << "\n " << k << " ";
    
    sdkResetTimer(&timer); sdkStartTimer(&timer);
    
    cout << " Module: " << module_for_psi() << endl;
    
    evolution_with_potential_dt();
    
    cout << " Potential energy: " << potential_energy() << endl;
    cout << " Kinetic energy for psi: " << kinetic_energy_for_psi() << endl;
    
    forward_legendre_transform();
    backward_legendre_transform();
    
    sdkStopTimer(&timer); cout << " GPU time: " << sdkGetAverageTimerValue(&timer)*1e-3 << endl;
  }

  cout << " === End of EvolutionCUDA test ===\n" << endl;
}

void EvolutionCUDA::forward_legendre_transform()
{
  setup_legendre_transform();

  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;
  const int m = theta.m + 1;
  
  const Complex one(1.0, 0.0);
  const Complex zero(0.0, 0.0);

  insist(hipblasZgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
		     n1*n2, m, n_theta, 
		     (const hipDoubleComplex *) &one,
		     (const hipDoubleComplex *) psi_dev, n1*n2,
		     (const hipDoubleComplex *) weight_legendre_dev, n_theta,
		     (const hipDoubleComplex *) &zero,
		     (hipDoubleComplex *) legendre_psi_dev, n1*n2) == HIPBLAS_STATUS_SUCCESS);
}

void EvolutionCUDA::backward_legendre_transform()
{
  setup_legendre_transform();

  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;
  const int m = theta.m + 1;
  
  const Complex one(1.0, 0.0);
  const Complex zero(0.0, 0.0);
  
  insist(hipblasZgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
		     n1*n2, n_theta, m,
		     (const hipDoubleComplex *) &one,
		     (const hipDoubleComplex *) legendre_psi_dev, n1*n2,
		     (const hipDoubleComplex *) legendre_dev, m,
		     (const hipDoubleComplex *) &zero,
		     (hipDoubleComplex *) psi_dev, n1*n2) == HIPBLAS_STATUS_SUCCESS);
}

void EvolutionCUDA::forward_fft_for_psi()
{ 
  setup_cufft_plan_for_psi();
  
  insist(hipfftExecZ2Z(cufft_plan_for_psi, (hipDoubleComplex *) psi_dev, (hipDoubleComplex *) psi_dev, 
		      HIPFFT_FORWARD) == HIPFFT_SUCCESS);

  checkCudaErrors(hipDeviceSynchronize());
}

void EvolutionCUDA::backward_fft_for_psi()
{
  setup_cufft_plan_for_psi();
 
  insist(hipfftExecZ2Z(cufft_plan_for_psi, (hipDoubleComplex *) psi_dev, (hipDoubleComplex *) psi_dev, 
		      HIPFFT_BACKWARD) == HIPFFT_SUCCESS);
  
  checkCudaErrors(hipDeviceSynchronize());
}

double EvolutionCUDA::kinetic_energy_for_psi()
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;

  forward_fft_for_psi();

  const double *w = theta.w;
  
  insist(work_dev);
  hipDoubleComplex *psi_tmp_dev = (hipDoubleComplex *) work_dev;
  
  const int n_threads = 512;
  const int n_blocks = number_of_blocks(n_threads, n1*n2);
  
  double sum = 0.0;
  for(int k = 0; k < n_theta; k++) {
    const hipDoubleComplex *psi_in_dev = (hipDoubleComplex *) psi_dev + k*n1*n2;

    _psi_times_kinitic_energy_<<<n_blocks, n_threads, (n1+n2)*sizeof(double)>>>
    ((Complex *) psi_tmp_dev, (const Complex *) psi_in_dev, 
       kinetic_1_dev, n1, kinetic_2_dev, n2);
    
    checkCudaErrors(hipDeviceSynchronize());
    
    Complex dot(0.0, 0.0);
    insist(hipblasZdotc(cublas_handle, n1*n2, psi_in_dev, 1, psi_tmp_dev, 1, (hipDoubleComplex *) &dot)
	   == HIPBLAS_STATUS_SUCCESS);
    
    sum += w[k]*dot.real();
  }

  sum *= r1.dr*r2.dr/n1/n2;
  
  backward_fft_for_psi();
  
  const double s = 1.0/(n1*n2);
  insist(hipblasZdscal(cublas_handle, n1*n2*n_theta, &s, (hipDoubleComplex *) psi_dev, 1) 
	 == HIPBLAS_STATUS_SUCCESS);

  return sum;
}
