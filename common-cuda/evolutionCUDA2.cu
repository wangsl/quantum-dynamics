#include "hip/hip_runtime.h"

#include "evolutionCUDA.h"
#include "cumath.h"

__constant__ __device__ double dt;

inline int number_of_blocks(const int n_threads, const int n)
{ return n/n_threads*n_threads == n ? n/n_threads : n/n_threads+1; }

static __global__ void _evolution_with_potential_dt_(Complex *psi, const double *pot, int n)
{
  const int j = threadIdx.x + blockDim.x*blockIdx.x;
  if(j < n) psi[j] *= exp(Complex(0.0, -dt)*pot[j]);
}

void EvolutionCUDA::allocate_device_memories()
{ 
  cout << " Allocate device memory" << endl;
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;

  const int n = n1*n2*n_theta;
  
  cout << n1 << " " << n2 << " " << n_theta << " " << n << endl;
  
  // time step
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dt), &time.time_step, sizeof(double)));
  
  if(!pot_dev) {
    checkCudaErrors(hipMalloc(&pot_dev, n*sizeof(double)));
    insist(pot);
    checkCudaErrors(hipMemcpy(pot_dev, pot, n*sizeof(double), hipMemcpyHostToDevice));
  }

  if(!psi_dev) {
    checkCudaErrors(hipMalloc(&psi_dev, n*sizeof(Complex)));
    insist(psi);
    checkCudaErrors(hipMemcpy(psi_dev, psi, n*sizeof(Complex), hipMemcpyHostToDevice));
  }
  
  if(!work_dev) {
    const int max_dim = n1*n2 + n_theta + 1024;
    checkCudaErrors(hipMalloc(&work_dev, max_dim*sizeof(Complex)));
  }
  
  if(!w_dev) {
    checkCudaErrors(hipMalloc(&w_dev, n_theta*sizeof(double)));
    const double *w = theta.w;
    insist(w);
    checkCudaErrors(hipMemcpy(w_dev, w, n_theta*sizeof(double), hipMemcpyHostToDevice));
  }

  setup_cublas_handle();
  setup_cufft_plan();
}

void EvolutionCUDA::deallocate_device_memories()
{
  cout << " Deallocate device memory" << endl;
  
  if(pot_dev) { checkCudaErrors(hipFree(pot_dev)); pot_dev = 0; }
  if(psi_dev) { checkCudaErrors(hipFree(psi_dev)); psi_dev = 0; }
  if(work_dev) { checkCudaErrors(hipFree(work_dev)); work_dev = 0; }
  if(w_dev) { checkCudaErrors(hipFree(w_dev)); w_dev = 0; }
  if(exp_ipot_dt_dev) { checkCudaErrors(hipFree(exp_ipot_dt_dev)); exp_ipot_dt_dev = 0; }

  destroy_cublas_handle();
  destroy_cufft_plan();
}

void EvolutionCUDA::setup_cublas_handle()
{
  if(has_cublas_handle) return;
  insist(hipblasCreate(&cublas_handle) == HIPBLAS_STATUS_SUCCESS);
  has_cublas_handle = 1;
}

void EvolutionCUDA::destroy_cublas_handle()
{
  if(!has_cublas_handle) return;
  insist(hipblasDestroy(cublas_handle) == HIPBLAS_STATUS_SUCCESS);
  has_cublas_handle = 0;
}

void EvolutionCUDA::setup_cufft_plan()
{
  if(has_cufft_plan) return;

  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;
  
  int dim[] = { n1, n2 };

  insist(hipfftPlanMany(&cufft_plan, 2, dim, NULL, 1, n1*n2, NULL, 1, n1*n2,
		       HIPFFT_Z2Z, n_theta) == HIPFFT_SUCCESS);
  has_cufft_plan = 1;
}

void EvolutionCUDA::destroy_cufft_plan()
{
  if(!has_cufft_plan) return;
  insist(hipfftDestroy(cufft_plan) == HIPFFT_SUCCESS);
  has_cufft_plan = 0;
}

void EvolutionCUDA::evolution_with_potential_dt()
{
  insist(pot_dev && psi_dev);
  
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;
  const int n = n1*n2*n_theta;
  
  const int n_threads = 1024;
  const int n_blocks = number_of_blocks(n_threads, n);
  
  _evolution_with_potential_dt_<<<n_blocks, n_threads>>>(psi_dev, pot_dev, n);
}

double EvolutionCUDA::potential_energy()
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;

  const double *w = theta.w;
  
  insist(work_dev);
  hipDoubleComplex *psi_tmp_dev = (hipDoubleComplex *) work_dev;
  
  const int n_threads = 256;
  const int n_blocks = number_of_blocks(n_threads, n1*n2);
  
  double sum = 0.0;
  for(int k = 0; k < n_theta; k++) {
    const hipDoubleComplex *psi_in_dev = (hipDoubleComplex *) psi_dev + k*n1*n2;
    
    _vector_multiplication_<Complex, Complex, double><<<n_blocks, n_threads>>>
      ((Complex *) psi_tmp_dev, (const Complex *) psi_in_dev, pot_dev+k*n1*n2, n1*n2);
    
    checkCudaErrors(hipDeviceSynchronize());
    
    Complex dot(0.0, 0.0);
    insist(hipblasZdotc(cublas_handle, n1*n2, psi_in_dev, 1, psi_tmp_dev, 1, (hipDoubleComplex *) &dot)
	   == HIPBLAS_STATUS_SUCCESS);
    
    sum += w[k]*dot.real();
  }

  sum *= r1.dr*r2.dr;
  return sum;
}

double EvolutionCUDA::module_for_psi() const
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;
  
  const double *w = theta.w;
  
  double sum= 0.0;
  for(int k = 0; k < n_theta; k++) {
    Complex dot(0.0, 0.0);
    const hipDoubleComplex *psi_ = (hipDoubleComplex *) psi_dev + k*n1*n2;
    insist(hipblasZdotc(cublas_handle, n1*n2, psi_, 1, psi_, 1, (hipDoubleComplex *) &dot)
	   == HIPBLAS_STATUS_SUCCESS);
    sum += w[k]*dot.real();
  }

  sum *= r1.dr*r2.dr;
  return sum;
}

void EvolutionCUDA::cuda_fft_test()
{ 
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;
  
  insist(psi_dev);
  
  StopWatchInterface *timer = 0;
  sdkCreateTimer(&timer);

  const int &total_steps = time.total_steps;
  
  for(int k = 0; k < total_steps; k++) {
    
    cout << " " << k << " ";
    
    sdkResetTimer(&timer); sdkStartTimer(&timer);
    
    insist(hipfftExecZ2Z(cufft_plan, (hipDoubleComplex *) psi_dev, (hipDoubleComplex *) psi_dev, 
			HIPFFT_FORWARD) == HIPFFT_SUCCESS);
    
    insist(hipfftExecZ2Z(cufft_plan, (hipDoubleComplex *) psi_dev, (hipDoubleComplex *) psi_dev,
			HIPFFT_BACKWARD) == HIPFFT_SUCCESS);
    
    insist(hipDeviceSynchronize() == HIPFFT_SUCCESS);
    
    const double s = 1.0/(n1*n2);
    insist(hipblasZdscal(cublas_handle, n1*n2*n_theta, &s, (hipDoubleComplex *) psi_dev, 1) 
	   == HIPBLAS_STATUS_SUCCESS);
    
    sdkStopTimer(&timer); cout << "GPU time: " << sdkGetAverageTimerValue(&timer)*1e-3 << endl;
  }
}
