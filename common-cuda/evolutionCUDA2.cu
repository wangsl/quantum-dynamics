#include "hip/hip_runtime.h"

#include "evolutionCUDA.h"
#include "cumath.h"
#include "gradients.h"

void cuda_test();

// #define IDX2F(i,j,ld) ((((j)-1)*(ld))+((i)-1)) 1-based
// #define IDX2C(i,j,ld) (((j)*(ld))+(i)) 0-based

struct RadialCoordinates
{ 
  double dr;
  double r_left;
  double mass;
  int n;
};

__constant__ RadialCoordinates r1_dev;
__constant__ RadialCoordinates r2_dev;
__constant__ double dump1_dev[1024];
__constant__ double dump2_dev[1024];
__constant__ double energies_dev[1024];
__constant__ double legendre_weight_dev[256];

inline int number_of_blocks(const int n_threads, const int n)
{ return n/n_threads*n_threads == n ? n/n_threads : n/n_threads+1; }

inline void copy_radial_coordinates_to_device(const RadialCoordinates &r, const int n, 
					      const double dr, const double r_left, const double mass)	    
{									
  RadialCoordinates r_h;
  r_h.dr = dr;
  r_h.r_left = r_left;
  r_h.mass = mass;
  r_h.n = n;
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(r), &r_h, sizeof(RadialCoordinates))); 
}

static __global__ void _evolution_with_potential_(Complex *psi, const double *pot, int n, const double dt)
{
  const int index = threadIdx.x + blockDim.x*blockIdx.x;
  if(index < n) psi[index] *= exp(Complex(0.0, -dt)*pot[index]);
}

static __global__ void _evolution_with_kinetic_(Complex *psi, const int n1, const int n2, const int m, 
						const double dt)
{
  extern __shared__ double s_data[];
  
  double *kin1 = (double *) s_data;
  double *kin2 = (double *) &kin1[n1];
  
  cumath::setup_kinetic_energy_for_fft(kin1, r1_dev.n, r1_dev.n*r1_dev.dr, r1_dev.mass);
  cumath::setup_kinetic_energy_for_fft(kin2, r2_dev.n, r2_dev.n*r2_dev.dr, r2_dev.mass);
  __syncthreads();

  const int index = threadIdx.x + blockDim.x*blockIdx.x;
  if(index < n1*n2*m) {
    int i = -1; int j = -1; int k = -1;
    cumath::index_2_ijk(index, n1, n2, m, i, j, k);
    psi[index] *= exp(Complex(0.0, -dt)*(kin1[i]+kin2[j]));
  }
}

static __global__ void _evolution_with_rotational_(Complex *psi, const int n1, const int n2, const int m,
						   const double dt)
{
  extern __shared__ double s_data[];
  
  double *I1 = (double *) s_data;
  double *I2 = (double *) &I1[n1];

  cumath::setup_moments_of_inertia(I1, r1_dev.n, r1_dev.r_left, r1_dev.dr, r1_dev.mass);
  cumath::setup_moments_of_inertia(I2, r2_dev.n, r2_dev.r_left, r2_dev.dr, r2_dev.mass);
  __syncthreads();
  
  const int index = threadIdx.x + blockDim.x*blockIdx.x;
  if(index < n1*n2*m) {
    int i = -1; int j = -1; int l = -1;
    cumath::index_2_ijk(index, n1, n2, m, i, j, l);
    psi[index] *= exp(-Complex(0.0, 1.0)*dt*l*(l+1)*(I1[i]+I2[j]));
  }
}

static __global__ void _psi_times_kinitic_energy_(Complex *psiOut, const Complex *psiIn, 
						  const int n1, const int n2)
{
  extern __shared__ double s_data[];

  double *kin1 = (double *) s_data;
  double *kin2 = (double *) &kin1[n1];
  
  cumath::setup_kinetic_energy_for_fft(kin1, r1_dev.n, r1_dev.n*r1_dev.dr, r1_dev.mass);
  cumath::setup_kinetic_energy_for_fft(kin2, r2_dev.n, r2_dev.n*r2_dev.dr, r2_dev.mass);
  __syncthreads();

  const int index = threadIdx.x + blockDim.x*blockIdx.x;
  if(index < n1*n2) {
    int i = -1; int j = -1;
    cumath::index_2_ij(index, n1, n2, i, j);
    psiOut[index] = psiIn[index]*(kin1[i] + kin2[j]);
  }
}

static __global__ void _legendre_psi_times_moments_of_inertia_(Complex *psiOut, const Complex *psiIn, 
							      const int n1, const int n2)
{
  extern __shared__ double s_data[];

  double *I1 = (double *) s_data;
  double *I2 = (double *) &I1[n1];
  
  cumath::setup_moments_of_inertia(I1, r1_dev.n, r1_dev.r_left, r1_dev.dr, r1_dev.mass);
  cumath::setup_moments_of_inertia(I2, r2_dev.n, r2_dev.r_left, r2_dev.dr, r2_dev.mass);
  __syncthreads();
  
  const int index = threadIdx.x + blockDim.x*blockIdx.x;
  if(index < n1*n2) {
    int i = -1; int j = -1;
    cumath::index_2_ij(index, n1, n2, i, j);
    psiOut[index] = psiIn[index]*(I1[i] + I2[j]);
  }
}

static __global__ void _dump_wavepacket_(Complex *psi, const int n1, const int n2, const int n_theta)
{
  const int index = threadIdx.x + blockDim.x*blockIdx.x;
  if(index < n1*n2*n_theta) {
    int i = -1; int j = -1; int k = -1;
    cumath::index_2_ijk(index, n1, n2, n_theta, i, j, k);
    psi[index] *= dump1_dev[i]*dump2_dev[j];
  }
}

static __global__ void _psi_time_to_fai_energy_on_surface_(const int n, const int nE,
							   const double t, const double dt,
							   Complex *psi, Complex *fai,
							   Complex *dpsi, Complex *dfai)
{
  extern __shared__ Complex exp_iet_dt[];
  
  for(int i = threadIdx.x; i < nE; i += blockDim.x) 
    exp_iet_dt[i] = exp(Complex(0.0, t)*energies_dev[i])*dt;
  __syncthreads();

  const int index = threadIdx.x + blockDim.x*blockIdx.x;
  
  if(index < n*nE) {
    int i = -1; int iE = -1;
    cumath::index_2_ij(index, n, nE, i, iE);
    fai[index] += exp_iet_dt[iE] * psi[index];
    dfai[index] += exp_iet_dt[iE] * dpsi[index];
  }
}

static __global__ void _fai_on_dividing_surface_with_legendre_weight_(const int n1, const int n_theta,
								      const int nE, 
								      Complex *fai, const int op)
{
  const int index = threadIdx.x + blockDim.x*blockIdx.x;
  if(index < n1*n_theta*nE) {
    int i = -1; int k = -1; int iE = -1;
    cumath::index_2_ijk(index, n1, n_theta, nE, i, k, iE);
    if(op == 1)
      fai[index] *= legendre_weight_dev[k];
    else if(op == -1)
      fai[index] /= legendre_weight_dev[k];
  }
}

void gpu_memory_usage()
{
  size_t free_byte ;
  size_t total_byte ;
  checkCudaErrors(hipMemGetInfo(&free_byte, &total_byte));
  
  cout << " GPU memory usage:" 
       << " used = " << (total_byte-free_byte)/1024.0/1024.0 << "MB,"
       << " free = " << free_byte/1024.0/1024.0 << "MB,"
       << " total = " << total_byte/1024.0/1024.0 << "MB" <<endl;
}

void EvolutionCUDA::allocate_device_memories()
{ 
  cout << " Allocate device memory" << endl;

  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;

  const int n = n1*n2*n_theta;
  
  cout << " Wavepacket size: " << n1 << " " << n2 << " " << n_theta << " " << n << endl;
  
  if(!pot_dev) {
    checkCudaErrors(hipMalloc(&pot_dev, n*sizeof(double)));
    insist(pot);
    checkCudaErrors(hipMemcpy(pot_dev, pot, n*sizeof(double), hipMemcpyHostToDevice));
  }

  if(!psi_dev) {
    checkCudaErrors(hipMalloc(&psi_dev, n*sizeof(Complex)));
    insist(psi);
    checkCudaErrors(hipMemcpy(psi_dev, psi, n*sizeof(Complex), hipMemcpyHostToDevice));
  }
  
  if(!work_dev) {
    const int max_dim = n1*n2 + n_theta + 1024;
    checkCudaErrors(hipMalloc(&work_dev, max_dim*sizeof(Complex)));
  }
  
  if(!w_dev) {
    checkCudaErrors(hipMalloc(&w_dev, n_theta*sizeof(double)));
    const double *w = theta.w;
    insist(w);
    checkCudaErrors(hipMemcpy(w_dev, w, n_theta*sizeof(double), hipMemcpyHostToDevice));
  }

  if(apply_dump()) {
    size_t size = 0;
    
    checkCudaErrors(hipGetSymbolSize(&size, HIP_SYMBOL(dump1_dev)));
    insist(size/sizeof(double) > n1);
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dump1_dev), dump1.dump, n1*sizeof(double)));

    checkCudaErrors(hipGetSymbolSize(&size, HIP_SYMBOL(dump2_dev)));
    insist(size/sizeof(double) > n2);
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dump2_dev), dump2.dump, n2*sizeof(double)));
  }

  if(CRP.calculate_CRP) setup_CRP_data_on_device();

  copy_radial_coordinates_to_device(r1_dev, r1.n, r1.dr, r1.r[0], r1.mass);
  copy_radial_coordinates_to_device(r2_dev, r2.n, r2.dr, r2.r[0], r2.mass);

  setup_cublas_handle();
}

void EvolutionCUDA::deallocate_device_memories()
{
  cout << " Deallocate device memory" << endl;

#define _CUDA_FREE_(x) if(x) { checkCudaErrors(hipFree(x)); x = 0; }

  _CUDA_FREE_(pot_dev);
  _CUDA_FREE_(psi_dev);
  _CUDA_FREE_(work_dev);
  _CUDA_FREE_(w_dev);
  _CUDA_FREE_(legendre_dev);
  _CUDA_FREE_(weight_legendre_dev);
  _CUDA_FREE_(legendre_psi_dev);
  
#undef _CUDA_FREE_

  destroy_cublas_handle();
  destroy_cufft_plan_for_psi();
  destroy_cufft_plan_for_legendre_psi();
}

void EvolutionCUDA::setup_cublas_handle()
{
  if(has_cublas_handle) return;
  insist(hipblasCreate(&cublas_handle) == HIPBLAS_STATUS_SUCCESS);
  has_cublas_handle = 1;
}

void EvolutionCUDA::destroy_cublas_handle()
{
  if(!has_cublas_handle) return;
  insist(hipblasDestroy(cublas_handle) == HIPBLAS_STATUS_SUCCESS);
  has_cublas_handle = 0;
}

void EvolutionCUDA::setup_cufft_plan_for_psi()
{
  if(has_cufft_plan_for_psi) return;

  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;
  
  int dim[] = { n2, n1 };
  
  insist(hipfftPlanMany(&cufft_plan_for_psi, 2, dim, NULL, 1, n1*n2, NULL, 1, n1*n2,
		       HIPFFT_Z2Z, n_theta) == HIPFFT_SUCCESS);
  
  has_cufft_plan_for_psi = 1;
}

void EvolutionCUDA::destroy_cufft_plan_for_psi()
{
  if(!has_cufft_plan_for_psi) return;
  insist(hipfftDestroy(cufft_plan_for_psi) == HIPFFT_SUCCESS);
  has_cufft_plan_for_psi = 0;
}

void EvolutionCUDA::setup_cufft_plan_for_legendre_psi()
{
  if(has_cufft_plan_for_legendre_psi) return;
  
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int m = theta.m + 1;
  
  /* CUFFT performs FFTs in row-major or C order.
     For example, if the user requests a 3D transform plan for sizes X, Y, and Z,
     CUFFT transforms along Z, Y, and then X. 
     The user can configure column-major FFTs by simply changing the order of size parameters 
     to the plan creation API functions.
  */
  int dim[] = { n2, n1 };
  
  insist(hipfftPlanMany(&cufft_plan_for_legendre_psi, 2, dim, 
		       NULL, 1, 0, NULL, 1, 0, HIPFFT_Z2Z, m) == HIPFFT_SUCCESS);
  
  has_cufft_plan_for_legendre_psi = 1;
}

void EvolutionCUDA::destroy_cufft_plan_for_legendre_psi()
{
  if(!has_cufft_plan_for_legendre_psi) return;
  insist(hipfftDestroy(cufft_plan_for_legendre_psi) == HIPFFT_SUCCESS);
  has_cufft_plan_for_legendre_psi = 0;
}

void EvolutionCUDA::setup_legendre()
{
  if(legendre_dev) return;

  const int &n_theta = theta.n;
  const int m = theta.m + 1;
  const RMat &P = theta.legendre;

  Mat<Complex> P_complex(m, n_theta);
  for(int k = 0; k < n_theta; k++) {
    for(int l = 0; l < m; l++) {
      P_complex(l,k) = Complex(P(l,k), 0.0);
    }
  }
  
  checkCudaErrors(hipMalloc(&legendre_dev, m*n_theta*sizeof(Complex)));

  checkCudaErrors(hipMemcpy(legendre_dev, (const Complex *) P_complex,
			     m*n_theta*sizeof(Complex), hipMemcpyHostToDevice));
}

void EvolutionCUDA::setup_weight_legendre()
{ 
  if(weight_legendre_dev) return;

  const int &n_theta = theta.n;
  const int m = theta.m + 1;
  
  Mat<Complex> weight_legendre(n_theta, m);
  
  const double *w = theta.w;
  const RMat &P = theta.legendre;

  Mat<Complex> &wp = weight_legendre;
  
  for(int l = 0; l < m; l++) {
    const double f = l+0.5;
    for(int k = 0; k < n_theta; k++) {
      wp(k,l) = Complex(f*w[k]*P(l,k), 0.0);
    }
  }
  
  checkCudaErrors(hipMalloc(&weight_legendre_dev, m*n_theta*sizeof(Complex)));
  
  checkCudaErrors(hipMemcpy(weight_legendre_dev, (const Complex *) weight_legendre,
			     m*n_theta*sizeof(Complex), hipMemcpyHostToDevice));
  
}

void EvolutionCUDA::evolution_with_potential(const double dt)
{
  insist(pot_dev && psi_dev);
  
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;
  const int n = n1*n2*n_theta;
  
  const int n_threads = 512;
  const int n_blocks = number_of_blocks(n_threads, n);
  
  _evolution_with_potential_<<<n_blocks, n_threads>>>(psi_dev, pot_dev, n, dt);
}

double EvolutionCUDA::potential_energy()
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;

  const double *w = theta.w;
  
  insist(work_dev);
  hipDoubleComplex *psi_tmp_dev = (hipDoubleComplex *) work_dev;
  
  const int n_threads = 512;
  const int n_blocks = number_of_blocks(n_threads, n1*n2);
  
  double sum = 0.0;
  for(int k = 0; k < n_theta; k++) {
    const hipDoubleComplex *psi_in_dev = (hipDoubleComplex *) psi_dev + k*n1*n2;
    
    cumath::_vector_multiplication_<Complex, Complex, double><<<n_blocks, n_threads>>>
      ((Complex *) psi_tmp_dev, (const Complex *) psi_in_dev, pot_dev+k*n1*n2, n1*n2);
    
    checkCudaErrors(hipDeviceSynchronize());
    
    Complex dot(0.0, 0.0);
    insist(hipblasZdotc(cublas_handle, n1*n2, psi_in_dev, 1, psi_tmp_dev, 1, (hipDoubleComplex *) &dot)
	   == HIPBLAS_STATUS_SUCCESS);
    
    sum += w[k]*dot.real();
  }

  sum *= r1.dr*r2.dr;
  return sum;
}

double EvolutionCUDA::module_for_psi() const
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;
  
  const double *w = theta.w;
  
  double sum= 0.0;
  for(int k = 0; k < n_theta; k++) {
    Complex dot(0.0, 0.0);
    const hipDoubleComplex *psi_ = (hipDoubleComplex *) psi_dev + k*n1*n2;
    insist(hipblasZdotc(cublas_handle, n1*n2, psi_, 1, psi_, 1, (hipDoubleComplex *) &dot)
	   == HIPBLAS_STATUS_SUCCESS);
    sum += w[k]*dot.real();
  }

  sum *= r1.dr*r2.dr;
  return sum;
}

void EvolutionCUDA::setup_legendre_psi()
{
  if(legendre_psi_dev) return;

  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int m = theta.m + 1;
  
  checkCudaErrors(hipMalloc(&legendre_psi_dev, n1*n2*m*sizeof(Complex)));
  checkCudaErrors(hipMemset(legendre_psi_dev, 0, n1*n2*m*sizeof(Complex)));
}

void EvolutionCUDA::cuda_fft_test()
{ 
  cout << " === EvolutionCUDA test ===" << endl;

  time_evolution();
  
  cout << " === End of EvolutionCUDA test ===\n" << endl;
}

void EvolutionCUDA::time_evolution()
{
  insist(psi_dev);
  
  const int &total_steps = time.total_steps;
  int &steps = time.steps;
  const double &dt = time.time_step;
  
  StopWatchInterface *timer = 0;
  sdkCreateTimer(&timer);
  
  for(int k = 0; k < total_steps; k++) {
    
    cout << "\n Step: " << k << endl;

    sdkResetTimer(&timer); sdkStartTimer(&timer);

    if(k == 0 && steps == 0) evolution_with_potential(-dt/2);
    
    evolution_with_potential(dt);
    
    forward_legendre_transform();
    
    evolution_with_rotational(dt/2);
    
    forward_fft_for_legendre_psi();
    
    evolution_with_kinetic(dt);
    
    const double e_kin = kinetic_energy_for_legendre_psi(0);
    
    backward_fft_for_legendre_psi(1);
    
    evolution_with_rotational(dt/2);

    const double e_rot = rotational_energy(0);
    
    backward_legendre_transform();

    const double e_pot = potential_energy();
    const double module = module_for_psi();
    
    cout << " e_kin: " << e_kin << "\n"
	 << " e_rot: " << e_rot << "\n"
	 << " e_pot: " << e_pot << "\n"
	 << " e_tot: " << e_kin + e_rot + e_pot << "\n"
	 << " module: " << module << endl;

    steps++;
    
    dump_wavepacket();
    
    const int calculate_CRP = steps%options.steps_to_copy_psi_from_device_to_host == 0 ? 1 : 0;
    //calculate_reaction_probabilities(calculate_CRP, (k+1)*dt);

    if(options.wave_to_matlab && steps%options.steps_to_copy_psi_from_device_to_host == 0) {
      copy_psi_from_device_to_host();
      wavepacket_to_matlab(options.wave_to_matlab);
    }
    
    sdkStopTimer(&timer); cout << " GPU time: " << sdkGetAverageTimerValue(&timer)*1e-3 << endl;

    cout.flush();
  }
}

void EvolutionCUDA::forward_legendre_transform()
{
  setup_legendre_transform();

  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;
  const int m = theta.m + 1;
  
  const Complex one(1.0, 0.0);
  const Complex zero(0.0, 0.0);

  insist(hipblasZgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
		     n1*n2, m, n_theta, 
		     (const hipDoubleComplex *) &one,
		     (const hipDoubleComplex *) psi_dev, n1*n2,
		     (const hipDoubleComplex *) weight_legendre_dev, n_theta,
		     (const hipDoubleComplex *) &zero,
		     (hipDoubleComplex *) legendre_psi_dev, n1*n2) == HIPBLAS_STATUS_SUCCESS);
}

void EvolutionCUDA::backward_legendre_transform()
{
  setup_legendre_transform();
  
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;
  const int m = theta.m + 1;
  
  const Complex one(1.0, 0.0);
  const Complex zero(0.0, 0.0);

  insist(hipblasZgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
		     n1*n2, n_theta, m,
		     (const hipDoubleComplex *) &one,
		     (const hipDoubleComplex *) legendre_psi_dev, n1*n2,
		     (const hipDoubleComplex *) legendre_dev, m,
		     (const hipDoubleComplex *) &zero,
		     (hipDoubleComplex *) psi_dev, n1*n2) == HIPBLAS_STATUS_SUCCESS);
}

void EvolutionCUDA::forward_fft_for_psi()
{ 
  setup_cufft_plan_for_psi();
  
  insist(hipfftExecZ2Z(cufft_plan_for_psi, (hipDoubleComplex *) psi_dev, (hipDoubleComplex *) psi_dev, 
		      HIPFFT_FORWARD) == HIPFFT_SUCCESS);
}

void EvolutionCUDA::backward_fft_for_psi(const int do_scale)
{
  setup_cufft_plan_for_psi();
  
  insist(hipfftExecZ2Z(cufft_plan_for_psi, (hipDoubleComplex *) psi_dev, (hipDoubleComplex *) psi_dev, 
		      HIPFFT_BACKWARD) == HIPFFT_SUCCESS);
  
  if(do_scale) {
    const int &n1 = r1.n;
    const int &n2 = r2.n;
    const int &n_theta = theta.n;
    const double s = 1.0/(n1*n2);
    insist(hipblasZdscal(cublas_handle, n1*n2*n_theta, &s, (hipDoubleComplex *) psi_dev, 1) 
	   == HIPBLAS_STATUS_SUCCESS);
    
  }
}

double EvolutionCUDA::kinetic_energy_for_psi(const int do_fft)
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;
  
  if(do_fft) forward_fft_for_psi();
  
  const double *w = theta.w;
  
  insist(work_dev);
  hipDoubleComplex *psi_tmp_dev = (hipDoubleComplex *) work_dev;
  
  const int n_threads = 512;
  const int n_blocks = number_of_blocks(n_threads, n1*n2);
  
  double sum = 0.0;
  for(int k = 0; k < n_theta; k++) {
    const hipDoubleComplex *psi_in_dev = (hipDoubleComplex *) psi_dev + k*n1*n2;
    
    _psi_times_kinitic_energy_<<<n_blocks, n_threads, (n1+n2)*sizeof(double)>>>
      ((Complex *) psi_tmp_dev, (const Complex *) psi_in_dev, n1, n2);
    
    checkCudaErrors(hipDeviceSynchronize());
    
    Complex dot(0.0, 0.0);
    insist(hipblasZdotc(cublas_handle, n1*n2, psi_in_dev, 1, psi_tmp_dev, 1, (hipDoubleComplex *) &dot)
	   == HIPBLAS_STATUS_SUCCESS);
    
    sum += w[k]*dot.real();
  }
  
  sum *= r1.dr*r2.dr/n1/n2;
  
  if(do_fft) backward_fft_for_psi(1);
  
  return sum;
}

void EvolutionCUDA::forward_fft_for_legendre_psi()
{ 
  setup_cufft_plan_for_legendre_psi();
  
  insist(hipfftExecZ2Z(cufft_plan_for_legendre_psi, 
		      (hipDoubleComplex *) legendre_psi_dev, (hipDoubleComplex *) legendre_psi_dev, 
		      HIPFFT_FORWARD) == HIPFFT_SUCCESS);

  checkCudaErrors(hipDeviceSynchronize());
}

void EvolutionCUDA::backward_fft_for_legendre_psi(const int do_scale)
{
  setup_cufft_plan_for_legendre_psi();
  
  insist(hipfftExecZ2Z(cufft_plan_for_legendre_psi, 
		      (hipDoubleComplex *) legendre_psi_dev, (hipDoubleComplex *) legendre_psi_dev, 
		      HIPFFT_BACKWARD) == HIPFFT_SUCCESS);
  
  checkCudaErrors(hipDeviceSynchronize());
  
  if(do_scale) {
    const int &n1 = r1.n;
    const int &n2 = r2.n;
    const int m = theta.m + 1;
    
    const double s = 1.0/(n1*n2);
    insist(hipblasZdscal(cublas_handle, n1*n2*m, &s, (hipDoubleComplex *) legendre_psi_dev, 1) 
	   == HIPBLAS_STATUS_SUCCESS);
  }
}

double EvolutionCUDA::kinetic_energy_for_legendre_psi(const int do_fft)
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int m = theta.m + 1;
  
  if(do_fft) forward_fft_for_legendre_psi();

  insist(work_dev);
  hipDoubleComplex *psi_tmp_dev = (hipDoubleComplex *) work_dev;
  
  const int n_threads = 512;
  const int n_blocks = number_of_blocks(n_threads, n1*n2);
  
  double sum = 0.0;
  for(int l = 0; l < m; l++) {
    const hipDoubleComplex *legendre_psi_in_dev = (hipDoubleComplex *) legendre_psi_dev + l*n1*n2;
    
    _psi_times_kinitic_energy_<<<n_blocks, n_threads, (n1+n2)*sizeof(double)>>>
      ((Complex *) psi_tmp_dev, (const Complex *) legendre_psi_in_dev, n1, n2);
    
    checkCudaErrors(hipDeviceSynchronize());
    
    Complex dot(0.0, 0.0);
    insist(hipblasZdotc(cublas_handle, n1*n2, legendre_psi_in_dev, 1, psi_tmp_dev, 1, (hipDoubleComplex *) &dot)
	   == HIPBLAS_STATUS_SUCCESS);
    
    sum += 2.0/(2*l+1)*dot.real();
  }

  sum *= r1.dr*r2.dr/n1/n2;

  if(do_fft) backward_fft_for_legendre_psi(1);
  
  return sum;
}

double EvolutionCUDA::rotational_energy(const int do_legendre_transform)
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int m = theta.m + 1;
  
  if(do_legendre_transform) forward_legendre_transform();
  
  insist(work_dev);
  hipDoubleComplex *psi_tmp_dev = (hipDoubleComplex *) work_dev;

  const int n_threads = 512;
  const int n_blocks = number_of_blocks(n_threads, n1*n2);
  
  double sum = 0.0;
  for(int l = 0; l < m; l++) {
    const hipDoubleComplex *legendre_psi_in_dev = (hipDoubleComplex *) legendre_psi_dev + l*n1*n2;
    
    _legendre_psi_times_moments_of_inertia_<<<n_blocks, n_threads, (n1+n2)*sizeof(double)>>>
      ((Complex *) psi_tmp_dev, (const Complex *) legendre_psi_in_dev, n1, n2);
    
    checkCudaErrors(hipDeviceSynchronize());
    
    Complex dot(0.0, 0.0);
    insist(hipblasZdotc(cublas_handle, n1*n2, legendre_psi_in_dev, 1, psi_tmp_dev, 1, (hipDoubleComplex *) &dot)
	   == HIPBLAS_STATUS_SUCCESS);
    
    sum += l*(l+1)/(l+0.5)*dot.real();
  }

  sum *= r1.dr*r2.dr;

  if(do_legendre_transform) backward_legendre_transform();
  
  return sum;
}

double EvolutionCUDA::module_for_legendre_psi()
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int m = theta.m + 1;
  
  forward_legendre_transform();

  const int n_threads = 512;
  const int n_blocks = number_of_blocks(n_threads, n1*n2);
  
  double sum = 0.0;
  for(int l = 0; l < m; l++) {
    const hipDoubleComplex *legendre_psi_in_dev = (hipDoubleComplex *) legendre_psi_dev + l*n1*n2;
    
    Complex dot(0.0, 0.0);
    insist(hipblasZdotc(cublas_handle, n1*n2, legendre_psi_in_dev, 1, legendre_psi_in_dev,
		       1, (hipDoubleComplex *) &dot) == HIPBLAS_STATUS_SUCCESS);
    
    sum += 2.0/(2*l+1)*dot.real();
  }

  sum *= r1.dr*r2.dr;

  backward_legendre_transform();

  return sum;
}

void EvolutionCUDA::evolution_with_kinetic(const double dt)
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int m = theta.m + 1;
  const int n = n1*n2*m;
  
  const int n_threads = 512;
  const int n_blocks = number_of_blocks(n_threads, n);
  
  _evolution_with_kinetic_<<<n_blocks, n_threads, (n1+n2)*sizeof(double)>>>
    (legendre_psi_dev, n1, n2, m, dt);
}

void EvolutionCUDA::evolution_with_rotational(const double dt)
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int m = theta.m + 1;
  const int n = n1*n2*m;
  
  const int n_threads = 512;
  const int n_blocks = number_of_blocks(n_threads, n);
  
  _evolution_with_rotational_<<<n_blocks, n_threads, (n1+n2)*sizeof(double)>>>
    (legendre_psi_dev, n1, n2, m, dt);
}

void EvolutionCUDA::copy_psi_from_device_to_host()
{
  cout << " Copy wavepacket from device to host" << endl;
  
  insist(psi && psi_dev);

  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;
  
  checkCudaErrors(hipMemcpy(psi, psi_dev, n1*n2*n_theta*sizeof(Complex), hipMemcpyDeviceToHost));
}

void EvolutionCUDA::dump_wavepacket()
{
  if(!apply_dump()) return;

  cout << " Dump wavepacket" << endl;

  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;

  const int n_threads = 1024;
  const int n_blocks = number_of_blocks(n_threads, n1*n2*n_theta);

  _dump_wavepacket_<<<n_blocks, n_threads>>>(psi_dev, n1, n2, n_theta);
}

void EvolutionCUDA::setup_CRP_data_on_device()
{
  if(psi_on_surface_dev && d_psi_on_surface_dev && 
     fai_on_surface_dev && d_fai_on_surface_dev) return;
  
  const int &n1 = r1.n;
  const int &n_theta = theta.n;
  const int &n_energies = CRP.n_energies;
  
  cout << " Allocate CRP data memory on device" << endl;
  
  size_t size = 0;

  checkCudaErrors(hipGetSymbolSize(&size, HIP_SYMBOL(legendre_weight_dev)));
  insist(size/sizeof(double) > n_theta);
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(legendre_weight_dev), theta.w, n_theta*sizeof(double)));

  checkCudaErrors(hipGetSymbolSize(&size, HIP_SYMBOL(energies_dev)));
  insist(size/sizeof(double) > CRP.n_energies);
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(energies_dev), (const double *) CRP.energies, 
				     CRP.n_energies*sizeof(double)));
  
  if(!psi_on_surface_dev) {
    checkCudaErrors(hipMalloc(&psi_on_surface_dev, n1*n_theta*sizeof(Complex)));
    insist(psi_on_surface_dev);
  }
  
  if(!d_psi_on_surface_dev) {
    checkCudaErrors(hipMalloc(&d_psi_on_surface_dev, n1*n_theta*sizeof(Complex)));
    insist(d_psi_on_surface_dev);
  }
  
  if(!fai_on_surface_dev) {
    checkCudaErrors(hipMalloc(&fai_on_surface_dev, n1*n_theta*n_energies*sizeof(Complex)));
    insist(fai_on_surface_dev);
    checkCudaErrors(hipMemset(fai_on_surface_dev, 0, n1*n_theta*n_energies*sizeof(Complex)));
  }
  
  if(!d_fai_on_surface_dev) {
    checkCudaErrors(hipMalloc(&d_fai_on_surface_dev, n1*n_theta*n_energies*sizeof(Complex)));
    insist(d_fai_on_surface_dev);
    checkCudaErrors(hipMemset(d_fai_on_surface_dev, 0, n1*n_theta*n_energies*sizeof(Complex)));
  }
}

void EvolutionCUDA::calculate_psi_gradient_on_dividing_surface()
{
  cout << " Calculate Psi gradients on dividing surface" << endl;

  setup_CRP_data_on_device();

  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;
  const double &dr2 = r2.dr;

  const int &n_dividing_surface = CRP.n_dividing_surface;
  
  // not sure why CRP.n_gradient_points always gives segmentation fault error
  // CRP.n_gradient_points;
  const int n_gradient_points = 11;
  insist(n_gradient_points == 11);

  const int n_threads = 256;
  const int n_blocks = number_of_blocks(n_threads, n1*n_theta);
  
  gradients_3d<Complex><<<n_blocks, n_threads>>>(n1, n2, n_theta, n_dividing_surface, dr2, psi_dev,
						 psi_on_surface_dev, d_psi_on_surface_dev,
						 n_gradient_points);
}

void EvolutionCUDA::psi_time_to_fai_energy_on_surface(const double t)
{
  cout << " Psi to Fai on dividing surface" << endl;

  const int &n1 = r1.n;
  const int &n_theta = theta.n;
  const int &n_energies = CRP.n_energies;
  const double &dt = time.time_step;

  const int n_threads = 512;
  const int n_blocks = number_of_blocks(n_threads, n1*n_theta*n_energies);

  _psi_time_to_fai_energy_on_surface_<<<n_blocks, n_threads, n_energies*sizeof(Complex)>>>
    (n1*n_theta, n_energies, t, dt, 
     psi_on_surface_dev, fai_on_surface_dev,
     d_psi_on_surface_dev, d_fai_on_surface_dev);
}

void EvolutionCUDA::_calculate_reaction_probabilities()
{
  const int &n1 = r1.n;
  const int &n_theta = theta.n;
  const double &dr1 = r1.dr;
  const double &mu2 = r2.mass;
  
  const int &n_energies = CRP.n_energies;
  RVec &crp = CRP.CRP;
  const RVec &eta_sq = CRP.eta_sq;
  
  const double dr1_mu2 = dr1/mu2;
  
  fai_on_dividing_surface_times_legendre_weight();  
  
  for(int iE = 0; iE < n_energies; iE++) {
    const Complex *fai_ = fai_on_surface_dev + iE*n1*n_theta;
    const Complex *dfai_ = d_fai_on_surface_dev + iE*n1*n_theta;
    Complex dot(0.0, 0.0);
    insist(hipblasZdotc(cublas_handle, n1*n_theta,
		       (hipDoubleComplex *) dfai_, 1, 
		       (hipDoubleComplex *) fai_, 1, 
		       (hipDoubleComplex *) &dot) == HIPBLAS_STATUS_SUCCESS);

    crp[iE] = dot.imag()/eta_sq[iE]*dr1_mu2;
  }
  
  fai_on_dividing_surface_divides_legendre_weight();
}

void EvolutionCUDA::calculate_reaction_probabilities(const int cal_CRP, const double time)
{
  cout << " Calculate reaction probabilities" << endl;

  setup_CRP_data_on_device();
  calculate_psi_gradient_on_dividing_surface();
  psi_time_to_fai_energy_on_surface(time);
  
  if(cal_CRP) 
    _calculate_reaction_probabilities();
}

void EvolutionCUDA::fai_on_dividing_surface_with_legendre_weight(const int op)
{
  insist(op == 1 || op == -1);
  
  const int &n1 = r1.n;
  const int &n_theta = theta.n;
  const int &n_energies = CRP.n_energies;
  
  const int n_threads = 512;
  const int n_blocks = number_of_blocks(n_threads, n1*n_theta*n_energies);
  
  _fai_on_dividing_surface_with_legendre_weight_<<<n_blocks, n_threads>>>
    (n1, n_theta, n_energies, fai_on_surface_dev, op);
}
