#include "hip/hip_runtime.h"

#include "timeEvolCUDA.h"
#include "cumath.h"

#if 0
__constant__ __device__ double dt;

__global__ void setup_exp_ipot_dt_on_device(Complex *exp_pot, const double *pot, int n)
{
  const int j = threadIdx.x + blockDim.x*blockIdx.x;
  if(j < n) 
    exp_pot[j] = exp(Complex(0.0, -dt)*pot[j]);
}
#endif

void TimeEvolutionCUDA::allocate_device_memories()
{ 
  cout << " Allocate device memory" << endl;
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;

  cout << n1 << " " << n2 << " " << n_theta << endl;

  // time step
  //checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dt), &time.time_step, sizeof(double)));

  if(!pot_dev) {
    checkCudaErrors(hipMalloc(&pot_dev, n1*n2*n_theta*sizeof(double)));
    insist(pot);
    checkCudaErrors(hipMemcpy(pot_dev, pot, n1*n2*n_theta*sizeof(double), hipMemcpyHostToDevice));
  }

  if(!psi_dev) {
    checkCudaErrors(hipMalloc(&psi_dev, n1*n2*n_theta*sizeof(Complex)));
    insist(psi);
    checkCudaErrors(hipMemcpy(psi_dev, psi, n1*n2*n_theta*sizeof(Complex), hipMemcpyHostToDevice));
  }
  
  if(!work_dev) {
    const int max_dim = max(n1*n2 + n_theta, 100);
    checkCudaErrors(hipMalloc(&work_dev, max_dim*sizeof(Complex)));
  }
  
  if(!w_dev) {
    checkCudaErrors(hipMalloc(&w_dev, n_theta*sizeof(double)));
    const double *w = theta.w;
    insist(w);
    checkCudaErrors(hipMemcpy(w_dev, w, n_theta*sizeof(double), hipMemcpyHostToDevice));
  }
}

void TimeEvolutionCUDA::deallocate_device_memories()
{
  cout << " Deallocate device memory" << endl;

  if(pot_dev) { checkCudaErrors(hipFree(pot_dev)); pot_dev = 0; }
  if(psi_dev) { checkCudaErrors(hipFree(psi_dev)); psi_dev = 0; }
  if(work_dev) { checkCudaErrors(hipFree(work_dev)); work_dev = 0; }
  if(w_dev) { checkCudaErrors(hipFree(w_dev)); w_dev = 0; }
  if(exp_ipot_dt_dev) { checkCudaErrors(hipFree(exp_ipot_dt_dev)); exp_ipot_dt_dev = 0; }
}

void TimeEvolutionCUDA::cuda_fft_test()
{ 
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;
  
  hipblasHandle_t cublas_handle;
  insist(hipblasCreate(&cublas_handle) == HIPBLAS_STATUS_SUCCESS);
  
  hipfftHandle cufft_plan;
  insist(hipfftPlan2d(&cufft_plan, n1, n2, HIPFFT_Z2Z) == HIPFFT_SUCCESS);

  insist(psi_dev);
  
  StopWatchInterface *timer = 0;
  sdkCreateTimer(&timer);

  const int &total_steps = time.total_steps;
  
  for(int k = 0; k < total_steps; k++) {

    cout << "\n " << k << " ";

    sdkResetTimer(&timer);
    sdkStartTimer(&timer);
    
    Complex dot(0.0, 0.0);
    assert(hipblasZdotc(cublas_handle, n1*n2*n_theta, (hipDoubleComplex *) psi_dev, 1, 
		       (hipDoubleComplex *) psi_dev, 1, (hipfftDoubleComplex *) &dot) 
	   == HIPBLAS_STATUS_SUCCESS);
    cout << dot << " ";
    
    for(int l = 0; l < n_theta; l++) {
      hipDoubleComplex *psi_ = (hipfftDoubleComplex *) psi_dev + l*n1*n2;
      insist(hipfftExecZ2Z(cufft_plan, psi_, psi_, HIPFFT_FORWARD) == HIPFFT_SUCCESS);
    }
    
    for(int l = 0; l < n_theta; l++) {
      hipDoubleComplex *psi_ = (hipfftDoubleComplex *) psi_dev + l*n1*n2;
      insist(hipfftExecZ2Z(cufft_plan, psi_, psi_, HIPFFT_BACKWARD) == HIPFFT_SUCCESS);
    }
    
    const double s = 1.0/(n1*n2);
    insist(hipblasZdscal(cublas_handle, n1*n2*n_theta, &s, (hipDoubleComplex *) psi_dev, 1) 
	   == HIPBLAS_STATUS_SUCCESS);

    sdkStopTimer(&timer);
    double reduceTime = sdkGetAverageTimerValue(&timer);
    
    cout << "GPU time: " << reduceTime*1e-3 << endl;
  }
    
  insist(hipfftDestroy(cufft_plan) == HIPFFT_SUCCESS);
  insist(hipblasDestroy(cublas_handle) ==  HIPBLAS_STATUS_SUCCESS);
}

void TimeEvolutionCUDA::cuda_fft_test_with_many_plan()
{ 
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;
  
  hipblasHandle_t cublas_handle;
  insist(hipblasCreate(&cublas_handle) == HIPBLAS_STATUS_SUCCESS);
  
  int dim[] = { n1, n2 };

  hipfftHandle cufft_plan;
  insist(hipfftPlanMany(&cufft_plan, 2, dim, NULL, 1, n1*n2, NULL, 1, n1*n2,
		       HIPFFT_Z2Z, n_theta) == HIPFFT_SUCCESS);
  
  insist(psi_dev);
  
  StopWatchInterface *timer = 0;
  sdkCreateTimer(&timer);

  const int &total_steps = time.total_steps;
  
  for(int k = 0; k < total_steps; k++) {

    cout << "\n " << k << " ";

    sdkResetTimer(&timer); sdkStartTimer(&timer);
    
    insist(hipfftExecZ2Z(cufft_plan, (hipDoubleComplex *) psi_dev, (hipDoubleComplex *) psi_dev, 
			HIPFFT_FORWARD) == HIPFFT_SUCCESS);
    
    insist(hipfftExecZ2Z(cufft_plan, (hipDoubleComplex *) psi_dev, (hipDoubleComplex *) psi_dev,
			HIPFFT_BACKWARD) == HIPFFT_SUCCESS);

    insist(hipDeviceSynchronize() == HIPFFT_SUCCESS);
   
    const double s = 1.0/(n1*n2);
    insist(hipblasZdscal(cublas_handle, n1*n2*n_theta, &s, (hipDoubleComplex *) psi_dev, 1) 
	   == HIPBLAS_STATUS_SUCCESS);
    
    sdkStopTimer(&timer); cout << "GPU time: " << sdkGetAverageTimerValue(&timer)*1e-3 << endl;

    
    sdkResetTimer(&timer); sdkStartTimer(&timer);
    cuda_psi_normal_test();
    sdkStopTimer(&timer); cout << "GPU reduction time: " << sdkGetAverageTimerValue(&timer)*1e-3 << endl;

    sdkResetTimer(&timer); sdkStartTimer(&timer);
    cuda_psi_normal_test_with_stream();
    sdkStopTimer(&timer); cout << "GPU reduction with stream time: " << sdkGetAverageTimerValue(&timer)*1e-3 << endl;

  }
    
  insist(hipfftDestroy(cufft_plan) == HIPFFT_SUCCESS);
  insist(hipblasDestroy(cublas_handle) ==  HIPBLAS_STATUS_SUCCESS);
}

void TimeEvolutionCUDA::cuda_psi_normal_test()
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;

  hipblasHandle_t cublas_handle;
  insist(hipblasCreate(&cublas_handle) == HIPBLAS_STATUS_SUCCESS);
  insist(hipblasSetPointerMode(cublas_handle, HIPBLAS_POINTER_MODE_DEVICE) == HIPBLAS_STATUS_SUCCESS);

  Complex *mod_dev = (Complex *) work_dev;
  
  StopWatchInterface *timer = 0;
  sdkCreateTimer(&timer);
  
  sdkResetTimer(&timer); sdkStartTimer(&timer);
  for(int k = 0; k < n_theta; k++) {
    hipDoubleComplex *psi_ = (hipDoubleComplex *) psi_dev + k*n1*n2;
    insist(hipblasZdotc(cublas_handle, n1*n2, psi_, 1, psi_, 1, (hipDoubleComplex *) &mod_dev[k])
	   == HIPBLAS_STATUS_SUCCESS);
  }
  sdkStopTimer(&timer); cout << "Reduction 1 GPU time: " << sdkGetAverageTimerValue(&timer)*1e-3 << endl;

  insist(hipblasDestroy(cublas_handle) == HIPBLAS_STATUS_SUCCESS);

  Complex dot(0.0, 0.0);
  const int n_threads = 64;
  
  sdkResetTimer(&timer); sdkStartTimer(&timer);
  checkCudaErrors(hipMemset(mod_dev+n_theta, 0, sizeof(Complex)));
  DotProduct<double, Complex, Complex>					\
    <<<n_theta/n_threads+1, n_threads, n_threads*sizeof(Complex)>>>(w_dev, mod_dev, mod_dev+n_theta, n_theta);
  dot.zero();
  checkCudaErrors(hipMemcpy(&dot, mod_dev+n_theta, sizeof(Complex), hipMemcpyDeviceToHost));
  sdkStopTimer(&timer); cout << "Reduction 2 GPU time: " << sdkGetAverageTimerValue(&timer)*1e-3 << endl;
  dot *= r1.dr*r2.dr;
  cout << dot << endl;
}

void TimeEvolutionCUDA::cuda_psi_normal_test_with_stream()
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;

  hipblasHandle_t cublas_handle;
  insist(hipblasCreate(&cublas_handle) == HIPBLAS_STATUS_SUCCESS);
  insist(hipblasSetPointerMode(cublas_handle, HIPBLAS_POINTER_MODE_DEVICE) == HIPBLAS_STATUS_SUCCESS);

  hipStream_t *streams = (hipStream_t *) malloc(n_theta*sizeof(hipStream_t));
  insist(streams);
  for(int k = 0; k < n_theta; k++) 
    checkCudaErrors(hipStreamCreate(&streams[k]));
		    
  Complex *mod_dev = (Complex *) work_dev;
  
  StopWatchInterface *timer = 0;
  sdkCreateTimer(&timer);
  
  sdkResetTimer(&timer); sdkStartTimer(&timer);
  for(int k = 0; k < n_theta; k++) {
    insist(hipblasSetStream(cublas_handle, streams[k]) == HIPBLAS_STATUS_SUCCESS);
    
    hipDoubleComplex *psi_ = (hipDoubleComplex *) psi_dev + k*n1*n2;
    insist(hipblasZdotc(cublas_handle, n1*n2, psi_, 1, psi_, 1, (hipDoubleComplex *) &mod_dev[k])
	   == HIPBLAS_STATUS_SUCCESS);
  }
  sdkStopTimer(&timer); cout << "Reduction stream 1 GPU time: " << sdkGetAverageTimerValue(&timer)*1e-3 << endl;
  
  for(int k = 0; k < n_theta; k++) 
    checkCudaErrors(hipStreamDestroy(streams[k]));

  insist(hipblasDestroy(cublas_handle) == HIPBLAS_STATUS_SUCCESS);

  Complex dot(0.0, 0.0);
  const int n_threads = 64;
  
  sdkResetTimer(&timer); sdkStartTimer(&timer);
  checkCudaErrors(hipMemset(mod_dev+n_theta, 0, sizeof(Complex)));
  DotProduct<double, Complex, Complex>
    <<<n_theta/n_threads+1, n_threads, n_threads*sizeof(Complex)>>>(w_dev, mod_dev, mod_dev+n_theta, n_theta);
  dot.zero();
  checkCudaErrors(hipMemcpy(&dot, mod_dev+n_theta, sizeof(Complex), hipMemcpyDeviceToHost));
  sdkStopTimer(&timer); cout << "Reduction stream 2 GPU time: " << sdkGetAverageTimerValue(&timer)*1e-3 << endl;
  dot *= r1.dr*r2.dr;
  cout << dot << endl;
}


